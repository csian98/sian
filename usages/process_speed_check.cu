#include "hip/hip_runtime.h"
/**
 * @File		process_speed_check.cu
 * @brief		single thread, multi thread and SIMT GPU
 * @author		Jeong Hoon (Sian) Choi
 * @version 	1.0.0
 * @date		2024-05-19
 */

/* Copyright (C)
 * 2024 - Jeong Hoon (Sian) Choi
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <utility>
#include <algorithm>
#include <numeric>

#include <random>
#include <cmath>

#include <string_view>
#include <string>
#include <vector>

#include <thread>
#include <mutex>
#include <shared_mutex>
#include <condition_variable>

#include "sian/timer.h"

#if _TARGET_OS == OS_WINDOWS

#elif _TARGET_OS == OS_LINUX

// void custom_terminate_fnct(void) {
//	exit(1);
// }

template <typename T>
bool check_matrix(const T* a, const T* b, const int n, const int m, const double tolerance = 1e-5) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < m; ++j) {
			int index = i * m + j;
			if (std::fabs(a[index] - b[index]) > tolerance) {
				std::cout << a[index] << " : " << b[index] << "[" << index << "]" <<std::endl;
				return false;
			}
		}
	}
	return true;
}

template <typename T>
void single_thread(const T* a, const T* b, T* c, const int n, const int m, const int k) {
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < m; ++j) {
			double value = 0;
			for (int l = 0; l < k; ++l) {
				value += a[i * k + l] * b[l * m + j];
			}
			c[i * m + j] += value;
		}
	}
}

template <typename T>
void multi_thread(const T* a, const T* b, T* c, const int n, const int m, const int k,
				  const int thread_index, const int thread_num) {
	const int tasks = std::ceil(static_cast<float>(m) / thread_num);
	for (int i = 0; i < n; ++i) {
		for (int j = thread_index * tasks; j < (thread_index + 1) * tasks; ++j) {
			double value = 0;
			for (int l = 0; l < k; ++l) {
				if (j < m)
					value += a[i * k + l] * b[l * m + j];
			}
			c[i * m + j] += value;
		}
	}
}

static const int block_size = 32;

template <typename T>
__global__ void cuda_kernel(const T* a, const T* b, T* c, const int n, const int m, const int k) {
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int local_col = threadIdx.x;
	int local_row = threadIdx.y;
	
	__shared__ T partial_a[block_size][block_size];
	__shared__ T partial_b[block_size][block_size];

	double value = 0;
	
	for (int blk = 0; blk < std::ceil(static_cast<float>(k) / block_size); ++blk) {
		int stride = blk * block_size;

	    if (row >= n || stride + local_col >= k)
			partial_a[local_row][local_col] = 0;
		else
			partial_a[local_row][local_col] = a[row * k + (stride + local_col)];

		if (col >= m || stride + local_row >= k)
			partial_b[local_col][local_row] = 0;	// transpose (bank-confilic minimize)
		else
			partial_b[local_col][local_row] = b[(stride + local_row) * m + col];	// transpose
			
		__syncthreads();

		for (int i = 0; i < block_size; ++i) {
			value += partial_a[local_row][i] * partial_b[local_col][i];	// partial_b transpose
		}
		__syncthreads();
	}
	if (row >= n || col >= m) return;
	
	c[m * row + col] = value;
}

int main(int argc, char* argv[]) {
//	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	sian::Timer timer(3);

	std::random_device rd;
	std::mt19937 engine(rd());
	std::uniform_real_distribution<double> distribution(-1.0, 1.0);
	
	const int n = 1024 * 4;
	const int k = 1024 * 4;
	const int m = 1024 * 4;

	double* a = new double[n * k];
	double* b = new double[k * m];
	double* c1 = new double[n * m];
	double* c2 = new double[n * m];
	double* c3 = new double[n * m];

	for (int i = 0; i < n * k; ++i) a[i] = distribution(engine);
	for (int i = 0; i < k * m; ++i) b[i] = distribution(engine);

	memset(c1, 0, sizeof(n * m));
	memset(c2, 0, sizeof(n * m));

	std::cout << "####\nMatrix Multiply Parallel Calculation\n####\n" << std::endl;
	
	timer[0].set_name("single thread");
	timer[0].start();
	single_thread(a, b, c1, n, m ,k);
	timer[0].stop();
	
	timer[1].set_name("multi threads");
	const auto thread_num = std::thread::hardware_concurrency();
	std::vector<std::thread> threads;
	timer[1].start();
	for (int i = 0; i < thread_num; ++i) {
		threads.emplace_back(&multi_thread<double>, a, b, c2, n, m, k, i, thread_num);
	}

	for (auto& thread : threads) thread.join();
	timer[1].stop();
	std::cout << "multi thread is correct : " << std::boolalpha << check_matrix(c1, c2, n, m) << std::endl;

	timer[2].set_name("cuda GPU SIMT");
	timer[2].start();

	dim3 grid_dim(std::ceil(static_cast<float>(m) / block_size),
				  std::ceil(static_cast<float>(n) / block_size));
	dim3 block_dim(block_size, block_size);
	double* d_a;
	double* d_b;
	double* d_c;
	hipMalloc(&d_a, sizeof(double) * n * k);
	hipMalloc(&d_b, sizeof(double) * k * m);
	hipMalloc(&d_c, sizeof(double) * n * m);

	hipMemcpy(d_a, a, sizeof(double) * n * k, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(double) * k * m, hipMemcpyHostToDevice);
	hipMemset(d_c, 0, sizeof(double) * n * k);

	cuda_kernel<double><<<grid_dim, block_dim>>>(d_a, d_b, d_c, n, m ,k);

	hipMemcpy(c3, d_c, sizeof(double) * n * m, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	hipDeviceSynchronize();
	timer[2].stop();
	std::cout << "cuda GPU SIMT is correct : " << std::boolalpha << check_matrix(c1, c3, n, m) << std::endl;
	
	std::cout << timer;

	delete[] c2;
	delete[] c1;
	delete[] b;
	delete[] a;

	return 0;
}

#endif // OS dependency
