#include "hip/hip_runtime.h"
/**
 * @file		sample.cu
 * @brief		
 * @author		Jeong Hoon (Sian) Choi
 * @version 	1.0.0
 * @date		2024-04-03
 */

#include "sample_cuda.h"

/* C & CPP */
/*
#ifdef __cplusplus
extern "C" {
#endif
*/

#if _TARGET_OS == OS_WINDOWS

#elif _TARGET_OS == OS_LINUX

/* Definition */

/* Cuda Global & Device definition */

__global__ void kernel(void) {
//
}

/* Cuda Host Difinition */

void kernel_call(void) {	
	/*
	  // heterogeneous computing
	int ngpus;
	hipGetDeviceCount(&ngpus);

	for (int device = 0; devcie < gpus; ++device) {
		hipSetDevice(device);
		//
	}
	*/
	// kernel<<<dim3_grid, dim3_blk, dynamic_shared_size, stream>>>();
	
	// cudaDeviceSyncronize();
}

#endif // OS dependency

/*
#ifdef __cplusplus
}
#endif
*/
